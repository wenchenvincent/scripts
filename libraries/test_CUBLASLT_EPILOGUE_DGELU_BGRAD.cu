#include <string>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <hipblas.h>

inline void check_cublas_(hipblasStatus_t status) {
    if ( status != HIPBLAS_STATUS_SUCCESS ) {
        printf("CUBLAS Error: " + std::string(cublasGetStatusString(status)));
    }
}

#define NVTE_CHECK_CUBLAS(ans) { check_cublas_(ans); }

int main() {
   
    float one = 1.0;
    float zero = 0.0;
    float beta = zero;

    int m = 4, n = 3, k = 2;

    float *A_h, *B_h, *D_h, *pre_gelu_h, *bias_grad_h;
    A_h = (float*)malloc(sizeof(float)*m*k);
    B_h = (float*)malloc(sizeof(float)*n*k);
    D_h = (float*)malloc(sizeof(float)*m*n);
    pre_gelu_h = (float*)malloc(sizeof(float)*m*n);
    bias_grad_h = (float*)malloc(sizeof(float)*m);

    for (int i=0;i<m*k;i++)
	    A_h[i] = 0.1;

    for (int i=0;i<n*k;i++)
	    B_h[i] = 0.1;

    for (int i=0;i<m*n;i++)
	    pre_gelu_h[i] = 0.2;

    float *A, *B, *D, *pre_gelu, *bias_grad;
    hipMalloc(&A, sizeof(float)*m*k);
    hipMalloc(&B, sizeof(float)*n*k);
    hipMalloc(&D, sizeof(float)*m*n);
    hipMalloc(&pre_gelu, sizeof(float)*m*n);
    hipMalloc(&bias_grad, sizeof(float)*m);

    int lda = m, ldb = k, ldd = m;
    hipblasOperation_t transa = HIPBLAS_OP_N; 
    hipblasOperation_t transb = HIPBLAS_OP_N;
    hipDataType A_type = HIP_R_32F;
    hipDataType B_type = HIP_R_32F;
    hipDataType D_type = HIP_R_32F;
    hipDataType bias_type = HIP_R_32F;

    hipMemcpy(A, A_h, sizeof(float)*m*k, hipMemcpyHostToDevice);
    hipMemcpy(B, B_h, sizeof(float)*n*k, hipMemcpyHostToDevice);
    hipMemcpy(pre_gelu, pre_gelu_h, sizeof(float)*m*n, hipMemcpyHostToDevice);

    void* workspace;
    size_t workspaceSize = sizeof(float)*2*m*k*n;
    hipMalloc(&workspace, workspaceSize);


    hipblasLtHandle_t handle;
    NVTE_CHECK_CUBLAS(hipblasLtCreate(&handle));

    hipblasLtMatmulDesc_t       operationDesc = nullptr;
    hipblasLtMatrixLayout_t     Adesc = nullptr, Bdesc = nullptr, Ddesc = nullptr;
    hipblasLtMatmulPreference_t preference = nullptr;
    int                             returnedResults = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult = {};
    hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DGELU_BGRAD;

    int64_t ld_gelumat = (int64_t) ldd;

    hipblasComputeType_t gemm_compute_type =  HIPBLAS_COMPUTE_32F;


    // Create matrix descriptors. Not setting any extra attributes.
    NVTE_CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&Adesc, A_type,
                                                 transa == HIPBLAS_OP_N ? m : k,
                                                 transa == HIPBLAS_OP_N ? k : m,
                                                 lda));
    NVTE_CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&Bdesc, B_type,
                                                 transb == HIPBLAS_OP_N ? k : n,
                                                 transb == HIPBLAS_OP_N ? n : k,
                                                 ldb));
    NVTE_CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&Ddesc, D_type, m, n, ldd));

    NVTE_CHECK_CUBLAS(hipblasLtMatmulDescCreate(&operationDesc, gemm_compute_type, HIP_R_32F));
    NVTE_CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA,
                                                     &transa, sizeof(transa)));
    NVTE_CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB,
                                                     &transb, sizeof(transb)));



    NVTE_CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(operationDesc,
                                                         HIPBLASLT_MATMUL_DESC_BIAS_POINTER,
                                                         &bias_grad, sizeof(bias_grad)));
    NVTE_CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(
                                operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER,
                                &pre_gelu, sizeof(pre_gelu)));
    NVTE_CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(operationDesc,
                                                         HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD,
                                                         &ld_gelumat, sizeof(ld_gelumat)));


    NVTE_CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(operationDesc,
                                                     HIPBLASLT_MATMUL_DESC_EPILOGUE,
                                                     &epilogue, sizeof(epilogue)));

    NVTE_CHECK_CUBLAS(hipblasLtMatmulPreferenceCreate(&preference));
    NVTE_CHECK_CUBLAS(hipblasLtMatmulPreferenceSetAttribute(
                            preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
                            &workspaceSize, sizeof(workspaceSize)));

    NVTE_CHECK_CUBLAS(hipblasLtMatmulAlgoGetHeuristic(handle, operationDesc, Adesc, Bdesc, Ddesc,
                                                     Ddesc, preference, 1, &heuristicResult,
                                                     &returnedResults));

    if (returnedResults == 0) throw std::runtime_error("Unable to find any suitable algorithms");

    // D = alpha * (A * B) + beta * C
    NVTE_CHECK_CUBLAS(hipblasLtMatmul(handle,
                                     operationDesc,
                                     static_cast<const void*>(&one),         /* alpha */
                                     A,                                      /* A */
                                     Adesc,
                                     B,                                      /* B */
                                     Bdesc,
                                     static_cast<const void*>(&beta),        /* beta */
                                     D,                                      /* C */
                                     Ddesc,
                                     D,                                      /* D */
                                     Ddesc,
                                     &heuristicResult.algo,                  /* algo */
                                     workspace,                              /* workspace */
                                     workspaceSize,
                                     0));                               /* stream */

    hipMemcpy(D_h, D, sizeof(float)*m*n, hipMemcpyDeviceToHost);
    hipMemcpy(bias_grad_h, bias_grad, sizeof(float)*m, hipMemcpyDeviceToHost);

    for (int i=0; i<m*n;i++)
	    printf("D[%d]=%f\n", i, D_h[i]);
    for (int i=0; i<m;i++)
	    printf("bias_grad[%d]=%f\n", i, bias_grad_h[i]);

    NVTE_CHECK_CUBLAS(hipblasLtMatmulPreferenceDestroy(preference));
    NVTE_CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(Ddesc));
    NVTE_CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(Bdesc));
    NVTE_CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(Adesc));
    NVTE_CHECK_CUBLAS(hipblasLtMatmulDescDestroy(operationDesc));

    free(A_h);
    free(B_h);
    free(D_h);
    free(pre_gelu_h);
    free(bias_grad_h);

    hipFree(A);
    hipFree(B);
    hipFree(D);
    hipFree(pre_gelu);
    hipFree(bias_grad);
    hipFree(workspace);

}
